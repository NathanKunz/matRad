/*
% %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
%
% Copyright 2019 the matRad development team.
%
% This file is part of the matRad project. It is subject to the license
% terms in the LICENSE file found in the top-level directory of this
% distribution and at https://github.com/e0404/matRad/LICENSES.txt. No part
% of the matRad project, including this file, may be copied, modified,
% propagated, or distributed except according to the terms contained in the
% LICENSE file.
%
% %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
*/

/*
Mex Function for Computing a sparse vector product with

compiling needs a matlab supported c/c++ compiler e.g. Microsoft Visual Studio C++ or MinGW64 and CUDA
compile with matlab: mexcuda matRad_cuSparse.cu
compile with matlab for debug: mexcuda -v -g matRad_cuSparse.cu use (https://de.mathworks.com/help/matlab/matlab_external/debugging-on-microsoft-windows-platforms.html) for DB on windows
compile from matRad_Root: mexcuda  -outdir 'gpuAcceleration/mex' 'gpuAcceleration/mex/matRad_cuSparse.cu'

run with matlab: [pr,ir,jc] = seperateSparse(sparseMatrix);
*/

// include matlabs api
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"

// include Cuda runtime and Cusparse
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>

// checks for simplifying cuda code
#define CHECK_CUDA(func)                                        \
    {                                                           \
        hipError_t status = (func);                            \
        if (status != hipSuccess)                              \
        {                                                       \
            mexPrintf("CUDA failed at %d line with error: %s (%d)\n", __LINE__, hipGetErrorString(status), status); \
            mexErrMsgIdAndTxt(errId, "Critical CUSPARSE ERROR"); \
        }                                                       \
    }

#define CHECK_CUSPARSE(func)                   \
    {                                          \
        hipsparseStatus_t status = (func);      \
        if (status != HIPSPARSE_STATUS_SUCCESS) \
        {                                      \
            mexPrintf("CUDA failed at %d line with error: %s (%d)\n", __LINE__, hipsparseGetErrorString(status), status); \
            mexErrMsgIdAndTxt(errId, "Critical CUSPARSE ERROR"); \
        }                                                       \
    }

/*
define input arguments for less confusion
*/
#define NROWS_A prhs[0]
#define NCOLS_A prhs[1]
#define NNZ_A prhs[2]
#define JC_A prhs[3] // column offset size cols + 1
#define IR_A prhs[4] // row index size nnz
#define PR_A prhs[5] // values size nnz
#define TRANS prhs[6] // transpose flag
#define X_B prhs[7] // input Vector

void mexFunction(
    int nlhs, mxArray *plhs[],
    int nrhs, const mxArray *prhs[])
{

    char const *const errId = "matRad:gpuAcceleration:cuSparse:InvalidInput";
    char const *const errMsg = "Invalid input to MEX file";

    /* 
    check input and output arguments
    */
    if (nrhs != 8) mexErrMsgIdAndTxt(errId, "Wrong number of input arguments");

    if (!mxIsScalar(NROWS_A)) mexErrMsgIdAndTxt(errId, "Argument 1 ROWS must be scalar");
    if (!mxIsScalar(NCOLS_A)) mexErrMsgIdAndTxt(errId, "Argument 2 COLS must be scalar");
    if (!mxIsScalar(NNZ_A)) mexErrMsgIdAndTxt(errId, "Argument 3 NNZ must be scalar");
    if (!mxIsScalar(TRANS)) mexErrMsgIdAndTxt(errId, "Argument 7 Transpose Flag must be scalar");

    if (!mxIsGPUArray(JC_A) && !mxGPUIsValidGPUData(JC_A)) mexErrMsgIdAndTxt(errId, "Argument 4 JC must be gpu array");
    if (!mxIsGPUArray(IR_A) && !mxGPUIsValidGPUData(IR_A)) mexErrMsgIdAndTxt(errId, "Argument 5 IR must be gpu array");
    if (!mxIsGPUArray(PR_A) && !mxGPUIsValidGPUData(PR_A)) mexErrMsgIdAndTxt(errId, "Argument 6 PR must be gpu array");
    if (!mxIsGPUArray(X_B) && !mxGPUIsValidGPUData(X_B)) mexErrMsgIdAndTxt(errId, "Argument 7 Vector B must be gpu array");

    // Initializie MathWorks Parallel Gpu API
    mxInitGPU();

    // Create read only pointer to gpu arrays
    mxGPUArray const *ir_a = mxGPUCreateFromMxArray(IR_A);
    mxGPUArray const *jc_a = mxGPUCreateFromMxArray(JC_A);
    mxGPUArray const *pr_a = mxGPUCreateFromMxArray(PR_A);

    mxGPUArray const *x = mxGPUCreateFromMxArray(X_B);

    mwSize A_n_rows = mxGetScalar(NROWS_A);
    mwSize A_n_cols = mxGetScalar(NCOLS_A);
    mwSize A_nnz = mxGetScalar(NNZ_A);

    mwSize *xdims = (mwSize*)mxGPUGetDimensions(x);

    if (mxGPUGetNumberOfDimensions(x) > 2) mexErrMsgIdAndTxt(errId, "Vector has to many dimensions");

    mwSize numelx = mxGPUGetNumberOfElements(x);
    hipsparseOperation_t trans = (hipsparseOperation_t)mxGetScalar(TRANS);
    //int nx = (trans == HIPSPARSE_OPERATION_NON_TRANSPOSE) ? xdims[0] : xdims[1];

    // check if size allows multiplication
    //mexPrintf("vector Dimensions x:%d y:%d \n", xdims[0], xdims[1]);
    //mexPrintf("numel in vector: %d\n, number of dimensions in vector: %d\n", mxGPUGetNumberOfElements(x), mxGPUGetNumberOfDimensions(x));
    //mexPrintf("A number cols: %d number row:%d \n", A_n_cols, A_n_rows);
    if (trans == HIPSPARSE_OPERATION_NON_TRANSPOSE)
    {
        if (numelx != A_n_cols)
            mexErrMsgIdAndTxt(errId, "Vector argument wrong size for multiply");
    }
    else
    {
        if (numelx != A_n_rows)
            mexErrMsgIdAndTxt(errId, "Vector argument wrong size for transpose multiply");
    }

    // check types
    if (mxGPUGetClassID(ir_a) != mxINT32_CLASS) mexErrMsgIdAndTxt(errId, "IR is not int32");
    if (mxGPUGetClassID(jc_a) != mxINT32_CLASS) mexErrMsgIdAndTxt(errId, "JC is not int32");
    if (mxGPUGetClassID(pr_a) != mxSINGLE_CLASS) mexErrMsgIdAndTxt(errId, "VAL is not single");
    if (mxGPUGetClassID(x) != mxSINGLE_CLASS) mexErrMsgIdAndTxt(errId, "Vector V is not single");

    // check complexity
    if (mxGPUGetComplexity(pr_a) != mxREAL) mexErrMsgIdAndTxt(errId, "Complex arguments are not supported");
    if (mxGPUGetComplexity(x) != mxREAL) mexErrMsgIdAndTxt(errId, "Complex arguments are not supported");


    // return vector
    const mwSize ndim = 1;
    mwSize dims[ndim] = { trans == HIPSPARSE_OPERATION_NON_TRANSPOSE ? A_n_rows : A_n_cols };
    mxClassID cid = mxGPUGetClassID(x);
    mxGPUArray* y;

    y = mxGPUCreateGPUArray(ndim, dims, cid, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (y == NULL) mexErrMsgIdAndTxt(errId, "mxGPUCreateGPUArray failed");

    // CUSPARSE APIs Y=α*op(A)⋅X+β*Y
    hipsparseHandle_t handle = NULL;
    hipsparseStatus_t status;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void* d_buffer = NULL;
    size_t bufferSize = 0;

    CHECK_CUSPARSE( hipsparseCreate(&handle) );

    // Convert matlab pointer to native pointer and types
    int* const d_ir_a = (int*)mxGPUGetDataReadOnly(ir_a); // data row index of a
    int* const d_jc_a = (int*)mxGPUGetDataReadOnly(jc_a); // data coloumn indexing of a
    float * const d_val = (float *)mxGPUGetDataReadOnly(pr_a); // data values of a
    float * const d_x = (float *)mxGPUGetDataReadOnly(x); // data in vector x
    float* d_y = (float *)mxGPUGetData(y); // data in (return) vector y
    float alpha = 1.0f;
    float beta = 0.0f;

    // create sparse matrix A
    CHECK_CUSPARSE( hipsparseCreateCsc(&matA, A_n_rows, A_n_cols, A_nnz, d_jc_a, d_ir_a, d_val, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) );

    // create dense vector x
    int x_numel = (trans == HIPSPARSE_OPERATION_NON_TRANSPOSE) ? A_n_cols: A_n_rows;
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, x_numel, d_x, HIP_R_32F) );

    // create dense output vector y
    int y_numel = (trans == HIPSPARSE_OPERATION_NON_TRANSPOSE) ? A_n_rows : A_n_cols;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, y_numel, d_y, HIP_R_32F));

        // create buffer if needed
    CHECK_CUSPARSE(
        hipsparseSpMV_bufferSize(
            handle, trans,
            &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
            HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));

    if (bufferSize > 0)
    {
        hipError_t status = hipMalloc(&d_buffer, bufferSize);
        if (status != hipSuccess)
            mexErrMsgIdAndTxt(errId, "Critical CUSPARSE ERROR");
    }

    // execute SpMV
    CHECK_CUSPARSE(hipsparseSpMV(handle, trans, &alpha, matA, vecX, &beta, vecY, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, d_buffer));

    /* return result this status check has some problems and return unrecognized error codes now and then propable when no status was set beforehand or another gpu operation writes into status in between operations
    if (status == HIPSPARSE_STATUS_SUCCESS)
    {
        plhs[0] = mxGPUCreateMxArrayOnGPU(y);
    }else
    {
        mexPrintf("CUDA failed at %d line with error: %s (%d)\n", __LINE__, hipsparseGetErrorString(status), status);
        mexErrMsgTxt("Unkown Error in cu sparse");
    }*/

    // return data
    plhs[0] = mxGPUCreateMxArrayOnGPU(y);

    // free data
    // destroy cuda matrix/ vector descriptors and buffer
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    if (d_buffer) CHECK_CUDA(hipFree(d_buffer));
    mxGPUDestroyGPUArray(ir_a);
    mxGPUDestroyGPUArray(pr_a);
    mxGPUDestroyGPUArray(jc_a);
    mxGPUDestroyGPUArray(x);
    mxGPUDestroyGPUArray(y);
    mxFree(xdims);

    return;
}